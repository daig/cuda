#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#define N 10000000

__global__ void vector_add(float *out, float *a, float *b, int n) {
    for (int i = 0; i < n; i++){ out[i] = a[i] + b[i]; }
}

int main() {
    hipCtx_t* ctx;
    cuCxtCreate(ctx, 0, 0);

}
