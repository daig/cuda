#include <hip/hip_runtime.h>
#include <iostream>
#include <string>
#include <vector>
#include "result.hxx"
// #include "context.hxx"
#include "version.hxx"
#include "device.hxx"
#include "context.hxx"
#include "module.hxx"

namespace cuda {
inline auto init() noexcept { return static_cast<result_t>(hipInit(0)); }
} // namespace cuda

using std::cout, std::endl;

int main(){

    cuda::init();
    cout << "CUDA Driver Version: " << cuda::driver_version() << endl;
    auto dev = cuda::device_t::get(0);
    cuda::context_t ctx(dev);

  const char* ptxCode =
    ".version 7.0\n"
    ".target sm_30\n"
    "\n"
    ".visible .func _Z6addIntii(\n"
    "  .param .s32 addend1,\n"
    "  .param .s32 addend2)\n"
    "{\n"
    "  .reg .s32 sum;\n"
    "  add.s32 sum, addend1, addend2;\n"
    "  ret.s32 sum;\n"
    "}\n";
  cuda::modul::option_list opts;
  using cujit = cuda::modul::jit_option;
  size_t buffer_size = 1024; char buffer[buffer_size];
  opts.info_log_buffer_size_bytes(buffer_size)
      .info_log_buffer(buffer);
  cuda::modul m(ptxCode, opts);
  auto f = m.get_function("addInt");

}

