#include <hip/hip_runtime.h>
#include <iostream>
#include <string>
#include <vector>
#include <fstream>
#include "result.hxx"
// #include "context.hxx"
#include "version.hxx"
#include "device.hxx"
#include "context.hxx"
#include "module.hxx"
#include "library.hxx"
#include "memory.hxx"
#include "stream.hxx"

using std::cout, std::endl;
namespace cuda {
inline auto init() noexcept { return static_cast<result_t::e>(hipInit(0)); }
} // namespace cuda


void* readFile(const char* filename) {

    using std::cout, std::endl;
    std::ifstream file(filename, std::ios::ate);
    if (!file) { std::cerr << "Failed to open file: " << filename << endl; return nullptr;}
    std::streamsize size = file.tellg();
    file.seekg(0, std::ios::beg);
    char* buffer = new char[size];
    if (!file.read(buffer, size)) { std::cerr << "Failed to read file: " << filename << endl; return nullptr;}

    file.close();
    return buffer;
}


int main(){

    cuda::init();
    std::cout << "CUDA Driver Version: " << cuda::driver_version() << std::endl;
    auto devices = cuda::device_t::all();
    auto dev = devices[0];
    cuda::context_t ctx(dev);

    const void* ptxCode = readFile("test.ptx");

  cuda::jit::option_list opts;
  using cujit = cuda::jit::option;
  size_t buffer_size = 1024; char buffer[buffer_size]{};
  opts.info_log_buffer_size_bytes(buffer_size)
      .info_log_buffer(buffer);
  cuda::modul m(ptxCode, opts);
  auto f = m.get_function("_Z3addPiS_S_");
  for (int i = 0; i < buffer_size; i++) { std::cout << buffer[i]; }
  cuda::result_t r = static_cast<cuda::result_t::e>(hipDeviceGetPCIBusId(buffer, buffer_size, dev.raw));
  cuda::device_t dev2(buffer);
  cout << "hipDeviceGetPCIBusId: " << r << endl;
  cout << buffer << endl;
  cout << (dev2 == dev) << endl;
}

