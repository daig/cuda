#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include <string>
#include "result.hxx"
// #include "context.hxx"
#include "version.hxx"
#include "device.hxx"
#include "context.hxx"
#include "module.hxx"

namespace cuda {
inline auto init() noexcept { return static_cast<result_t>(hipInit(0)); }
} // namespace cuda

using std::cout, std::endl;

int main(){

    cuda::init();
    cout << "CUDA Driver Version: " << cuda::driver_version() << endl;
    auto d = cuda::device_t::get(0);
    int x = d.get(cuda::device_t::attribute::max_threads_per_block);
    cout << "max threads per block: " << x << endl;

    cout << "affinity support: " << d.execAffinitySupport() << endl;
    cout << "total memory: " << d.totalMem() << endl;

    auto s = cuda::result::string(cuda::result_t::captured_event);
    cout << s << endl;
}

