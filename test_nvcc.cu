
#include <hip/hip_runtime.h>
#include <iostream>
// nvcc -keep -keep-dir test_output test0.cu
#define N 10000000

using namespace std;

__global__ void add(int *a, int *b, int *c) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < N) { c[tid] = a[tid] + b[tid]; }
}


int main() {
    int *a, *b, *c;
    int *d_a, *d_b, *d_c;
    int size = N * sizeof(int);

    hipMalloc((int **) &d_a, size);
    hipMalloc((int **) &d_b, size);
    hipMalloc((int **) &d_c, size);

    a = (int *) malloc(size);
    b = (int *) malloc(size);
    c = (int *) malloc(size);

    for (int i = 0; i < N; i++) {
        a[i] = i;
        b[i] = i;
    }

    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

    add<<<(N + 255) / 256, 256>>>(d_a, d_b, d_c);

    hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

    for (int i = 0; i < N; i++) {
        cout << c[i] << endl;
    }

    free(a);
    free(b);
    free(c);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0;

}

